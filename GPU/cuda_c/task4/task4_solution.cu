
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

__global__ void saxpy(float scalar, float * x, float * y)
{
    // Determine our unique global thread ID, so we know which element to process
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ( tid < N ) // Make sure we don't do more work than we have data!
        y[tid] = scalar * x[tid] + y[tid];
}

int main()
{
    float *x, *y;

    int size = N * sizeof (float); // The total number of bytes per vector

    hipError_t ierrAsync;
    hipError_t ierrSync;

    // Allocate memory
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int threads_per_block = 256;
    int number_of_blocks = (N / threads_per_block) + 1;

    saxpy <<< number_of_blocks, threads_per_block >>> ( 2.0f, x, y );

    ierrSync = hipGetLastError();
    ierrAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
    if (ierrSync != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierrSync)); }
    if (ierrAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(ierrAsync)); }

    // Print out our Max Error
    float maxError = 0;
    for( int i = 0; i < N; ++i )
        if (abs(4-y[i]) > maxError) { maxError = abs(4-y[i]); }
    printf("Max Error: %.5f", maxError);

    // Free all our allocated memory
    hipFree( x ); hipFree( y );
}