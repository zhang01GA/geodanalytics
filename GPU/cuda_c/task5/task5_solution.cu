
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NX 200
#define NY 100

__global__ void saxpy2D(float scalar, float * x, float * y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if ( row < NX && col < NY ) // Make sure we don't do more work than we have data!
        y[row*NY+col] = scalar * x[row*NY+col] + y[row*NY+col];
}

int main()
{
    float *x, *y;
    float maxError = 0;

    int size = NX * NY * sizeof (float); // The total number of bytes per vector

    hipError_t ierrAsync;
    hipError_t ierrSync;

    hipDeviceProp_t prop;

    // Allocate memory
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);

    // Initialize memory
    for( int i = 0; i < NX*NY; ++i )
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    dim3 threads_per_block (32,16,1);
    dim3 number_of_blocks ((NX/threads_per_block.x)+1,
                           (NY/threads_per_block.y)+1,
                           1);

    hipGetDeviceProperties(&prop, 0);
    if (threads_per_block.x * threads_per_block.y * threads_per_block.z > prop.maxThreadsPerBlock) {
        printf("Too many threads per block ... exiting\n");
        goto cleanup;
    }
    if (threads_per_block.x > prop.maxThreadsDim[0]) {
        printf("Too many threads in x-direction ... exiting\n");
        goto cleanup;
    }
    if (threads_per_block.y > prop.maxThreadsDim[1]) {
        printf("Too many threads in y-direction ... exiting\n");
        goto cleanup;
    }
    if (threads_per_block.z > prop.maxThreadsDim[2]) {
        printf("Too many threads in z-direction ... exiting\n");
        goto cleanup;
    }

    saxpy2D <<< number_of_blocks, threads_per_block >>> ( 2.0f, x, y );

    ierrSync = hipGetLastError();
    ierrAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
    if (ierrSync != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierrSync)); }
    if (ierrAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(ierrAsync)); }

    // Print out our Max Error
    for( int i = 0; i < NX*NY; ++i )
        if (abs(4-y[i]) > maxError) { maxError = abs(4-y[i]); }
    printf("Max Error: %.5f", maxError);

cleanup:
    // Free all our allocated memory
    hipFree( x ); hipFree( y );
}