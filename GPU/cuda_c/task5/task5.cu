
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NX 200
#define NY 100

__global__ void saxpy2D(float scalar, float * x, float * y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if ( row < NX && col < NY ) // Make sure we don't do more work than we have data!
        y[row*NY+col] = scalar * x[row*NY+col] + y[row*NY+col];
}

int main()
{
    float *x, *y;
    float maxError = 0;

    int size = NX * NY * sizeof (float); // The total number of bytes per vector

    hipError_t ierrAsync;
    hipError_t ierrSync;

    // Allocate memory
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);

    // Initialize memory
    for( int i = 0; i < NX*NY; ++i )
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    dim3 threads_per_block (32,16,1);
    dim3 number_of_blocks ((NX/threads_per_block.x)+1,
                           (NY/threads_per_block.y)+1,
                           1);

    saxpy2D <<< number_of_blocks, threads_per_block >>> ( 2.0f, x, y );

    ierrSync = hipGetLastError();
    ierrAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
    if (ierrSync != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierrSync)); }
    if (ierrAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(ierrAsync)); }

    // Print out our Max Error
    for( int i = 0; i < NX*NY; ++i )
        if (abs(4-y[i]) > maxError) { maxError = abs(4-y[i]); }
    printf("Max Error: %.5f", maxError);

cleanup:
    // Free all our allocated memory
    hipFree( x ); hipFree( y );
}