
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{
    printf("Hello from Thread %d in block %d\n", threadIdx.x, blockIdx.x);
}

int main()
{
    hello<<<1,1>>>();
    hipDeviceSynchronize(); // The above call is asynchronous, wait until it
                             // finishes before exiting the program!

    return 0;
}