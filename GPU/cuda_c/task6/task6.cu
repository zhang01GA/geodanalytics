
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

struct DataElement
{
  char *name;
  int value;
};

__global__ 
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d\n", elem->name, elem->value);

  elem->name[0] = 'd';
  elem->value++;
}

void launch(DataElement *elem) {
  DataElement *d_elem;
  char *d_name;

  int namelen = strlen(elem->name) + 1;

  // Allocate storage for struct and text
  hipMalloc(&d_elem, sizeof(DataElement));
  hipMalloc(&d_name, namelen);

  // Copy up each piece separately, including new “text” pointer value
  hipMemcpy(d_elem, elem, sizeof(DataElement), hipMemcpyHostToDevice);
  hipMemcpy(d_name, elem->name, namelen, hipMemcpyHostToDevice);
  hipMemcpy(&(d_elem->name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

  // Finally we can launch our kernel, but CPU & GPU use different copies of “elem”
  Kernel<<< 1, 1 >>>(d_elem);

  hipMemcpy(&(elem->value), &(d_elem->value), sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(elem->name, d_name, namelen, hipMemcpyDeviceToHost);

  hipFree(d_name);
  hipFree(d_elem);
}

int main(void)
{
  DataElement *e;
  e = (DataElement*)malloc(sizeof(DataElement));

  e->value = 10;
  e->name = (char*)malloc(sizeof(char) * (strlen("hello") + 1));
  strcpy(e->name, "hello");

  launch(e);

  printf("On host: name=%s, value=%d\n", e->name, e->value);

  free(e->name);
  free(e);

  hipDeviceReset();
}